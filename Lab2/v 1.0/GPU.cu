#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>

#include "GPU.cuh"
#include "Utils.h"

__global__ void kernel(float a, float* x, int incx, float* y, int incy, int threads_num, int n)
{
	int global_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (global_index >= threads_num)
		return;
	if (global_index * incx >= n || global_index * incy >= n)
		return;

	y[global_index * incy] = y[global_index * incy] + a * x[global_index * incx];
}



float* saxpy_gpu(int threads_num, int block_size, int n, float a, float* x, int incx, float* y, int incy, std::vector<double>* times)
{
	int num_blocks = threads_num / block_size;
	if (threads_num % block_size != 0)
		num_blocks++;
	
	float* x_gpu;
	float* y_gpu;

	hipError_t error;
	error = hipMalloc((void**)&x_gpu, n * sizeof(float));
	if (error)
		printf("hipMalloc [x] error : %i \n", error);

	error = hipMalloc((void**)&y_gpu, n * sizeof(float));
	if (error)
		printf("hipMalloc [y] error: %i \n", error);

	error = hipMemcpy(x_gpu, x, n * sizeof(float), hipMemcpyHostToDevice);
	if (error)
		printf("hipMemcpy [x] error: %i \n", error);

	error = hipMemcpy(y_gpu, y, n * sizeof(float), hipMemcpyHostToDevice);
	if (error)
		printf("hipMemcpy [y] error: %i \n", error);

	double t1, t2;
	t1 = omp_get_wtime();
	kernel <<< num_blocks, block_size >>> (a, x_gpu, incx, y_gpu, incy, threads_num, n);
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(error));
	}

	error = hipDeviceSynchronize();

	t2 = omp_get_wtime();
	if (times != nullptr)
		(*times).push_back(t2 - t1);

	if (error != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", error);
	}

	error = hipMemcpy(y, y_gpu, n * sizeof(float), hipMemcpyDeviceToHost);
	if (error)
		printf("hipMemcpy device to host error: %i \n", error);

	hipFree(x_gpu);
	hipFree(y_gpu);

	return y;
}


__global__ void kerneld(double a, double* x, int incx, double* y, int incy, int threads_num, int n)
{
	int global_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (global_index >= threads_num)
		return;
	if (global_index * incx >= n || global_index * incy >= n)
		return;

	y[global_index * incy] = y[global_index * incy] + a * x[global_index * incx];
}

double* daxpy_gpu(int threads_num, int block_size, int n, double a, double* x, int incx, double* y, int incy, std::vector<double>* times)
{
	int num_blocks = threads_num / block_size;
	if (threads_num % block_size != 0)
		num_blocks++;


	double* x_gpu;
	double* y_gpu;

	hipError_t error;
	error = hipMalloc((void**)&x_gpu, n * sizeof(double));
	if (error)
		printf("hipMalloc [x] error : %i \n", error);

	error = hipMalloc((void**)&y_gpu, n * sizeof(double));
	if (error)
		printf("hipMalloc [y] error: %i \n", error);

	error = hipMemcpy(x_gpu, x, n * sizeof(double), hipMemcpyHostToDevice);
	if (error)
		printf("hipMemcpy [x] error: %i \n", error);

	error = hipMemcpy(y_gpu, y, n * sizeof(double), hipMemcpyHostToDevice);
	if (error)
		printf("hipMemcpy [y] error: %i \n", error);

	double t1, t2;
	t1 = omp_get_wtime();
	kerneld <<< num_blocks, block_size >>> (a, x_gpu, incx, y_gpu, incy, threads_num, n);

	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "Kerneld launch failed: %s\n", hipGetErrorString(error));
	}

	error = hipDeviceSynchronize();
	t2 = omp_get_wtime();
	if (times != nullptr)
	{
		(*times).push_back(t2 - t1);
	}

	if (error != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kerneld!\n", error);
	}
	
	error = hipMemcpy(y, y_gpu, n * sizeof(double), hipMemcpyDeviceToHost);
	if (error)
		printf("hipMemcpy device to host error: %i \n", error);

	hipFree(x_gpu);
	hipFree(y_gpu);

	return y;
}

